#include "hip/hip_runtime.h"
#include "stdio.h"
#include "ass1_lib.h"
#include <math.h>
#include <algorithm>
#include "hip/hip_runtime_api.h"
#include "hipblas.h"
extern "C" {
#include "cblas.h"
//}

// Nat
void matmult_nat(int m, int n, int k, double * A, double * B, double * C){
	for(int i = 0; i < m;i++){
		for(int j = 0; j < n;j++){
			C[i*n + j] = 0;
			for(int l = 0; l < k;l++){
				C[i*n + j] += A[i*k + l]*B[l*n + j];
			}
		}
	}
}

// library implementation through cblas
void matmult_lib(int m, int n, int k, double * A, double * B, double * C){
  cblas_dgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans, m, n, k, 1.0 , A, k, B, n, 0.0, C, n);
//printMat(C,m,n);
}

// Sequential on the GPU
__host__
void matmult_gpu1(int m, int n, int k, double * A, double * B, double * C){

	double *d_A, *d_B, *d_C;
	hipMalloc(&d_A,n*m*sizeof(double));
	hipMalloc(&d_B,k*m*sizeof(double));
	hipMalloc(&d_C,n*k*sizeof(double));

 	hipMemcpy(d_A,A,n*m*sizeof(double), hipMemcpyHostToDevice);
 	hipMemcpy(d_B,B,k*m*sizeof(double), hipMemcpyHostToDevice);
	
	cudaSeq<<<1,1>>>(m,n,k,d_A,d_B,d_C);
	hipMemcpy(C,d_C,n*m*sizeof(double), hipMemcpyDeviceToHost);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

}

// Naive GPU - 1 thread per element in C
__host__
void matmult_gpu2(int m, int n, int k, double * A, double * B, double * C){
	int K = 16;
	int gridx = ceil(n*1.0/K);
	int gridy = ceil(m*1.0/K);
	double *d_A, *d_B, *d_C;
	hipMalloc(&d_A,k*m*sizeof(double));
	hipMalloc(&d_B,k*n*sizeof(double));
	hipMalloc(&d_C,n*m*sizeof(double));

 	hipMemcpy(d_A,A,k*m*sizeof(double), hipMemcpyHostToDevice);
 	hipMemcpy(d_B,B,k*n*sizeof(double), hipMemcpyHostToDevice);
	cudaPar<<<dim3(gridx,gridy),dim3(K,K)>>>(m,n,k,d_A,d_B,d_C);
	hipMemcpy(C,d_C,n*m*sizeof(double), hipMemcpyDeviceToHost);
/*	#ifndef __print
	#define __print 5
	printMat(C,m,n);
	#endif
*/
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

}

// Each thread does neighbouring fields
__host__
void matmult_gpu3(int m, int n, int k, double * A, double * B, double * C){
	int K = 16;
	int p = 2;
	int gridx = ceil(n*1.0/K);
	int gridy = ceil(m*1.0/K/p);
	double *d_A, *d_B, *d_C;

	checkCudaErrors(hipMalloc(&d_A,k*m*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_B,n*k*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_C,n*m*sizeof(double)));

 	checkCudaErrors(hipMemcpy(d_A,A,k*m*sizeof(double), hipMemcpyHostToDevice));
 	checkCudaErrors(hipMemcpy(d_B,B,k*n*sizeof(double), hipMemcpyHostToDevice));
	cudaPar2<<<dim3(gridx,gridy),dim3(K,K)>>>(m,n,k,p,d_A,d_B,d_C);
//	checkCudaErrors(hipDeviceSynchronize());
	hipMemcpy(C,d_C,n*m*sizeof(double), hipMemcpyDeviceToHost);
//	printMat(C,m,n);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}


void matmult_gpu4(int m, int n, int k, double * A, double * B, double * C){
	int K = 16;
	int p = 4;
	int gridx = ceil(n*1.0/K);
	int gridy = ceil(m*1.0/K/p);
	double *d_A, *d_B, *d_C;

	checkCudaErrors(hipMalloc(&d_A,k*m*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_B,n*k*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_C,n*m*sizeof(double)));

 	checkCudaErrors(hipMemcpy(d_A,A,k*m*sizeof(double), hipMemcpyHostToDevice));
 	checkCudaErrors(hipMemcpy(d_B,B,k*n*sizeof(double), hipMemcpyHostToDevice));
	cudaPar4<<<dim3(gridx,gridy),dim3(K,K)>>>(m,n,k,p,d_A,d_B,d_C);
//	checkCudaErrors(hipDeviceSynchronize());
	hipMemcpy(C,d_C,n*m*sizeof(double), hipMemcpyDeviceToHost);
//	printMat(C,m,n);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}


void matmult_gpu5(int m, int n, int k, double * A, double * B, double * C){
	
//	hipSetDevice(4);
	int K = 16;
	int gridx = floor(n*1.0/K);
	int gridy = floor(m*1.0/K);
	double *d_A, *d_B, *d_C;

	checkCudaErrors(hipMalloc(&d_A,k*m*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_B,n*k*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_C,n*m*sizeof(double)));

 	checkCudaErrors(hipMemcpy(d_A,A,k*m*sizeof(double), hipMemcpyHostToDevice));
 	checkCudaErrors(hipMemcpy(d_B,B,k*n*sizeof(double), hipMemcpyHostToDevice));
	cudaSMEM<<<dim3(gridx,gridy),dim3(K,K)>>>(m,n,k,d_A,d_B,d_C);
	hipDeviceSynchronize();
//	checkCudaErrors(hipDeviceSynchronize());
	hipMemcpy(C,d_C,n*m*sizeof(double), hipMemcpyDeviceToHost);
	//printMat(C,m,n);
	// printf("\n");

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}

void matmult_gpulib(int m, int n, int k, double * A, double * B, double * C){
	double alpha = 1.0;
	double beta = 0.0;
	const double *alphap, *betap;
	alphap = &alpha;
	betap = &beta;

	double *d_A, *d_B, *d_C;

	checkCudaErrors(hipMalloc(&d_A,k*m*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_B,n*k*sizeof(double)));
	checkCudaErrors(hipMalloc(&d_C,n*m*sizeof(double)));

 	checkCudaErrors(hipMemcpy(d_A,A,k*m*sizeof(double), hipMemcpyHostToDevice));
 	checkCudaErrors(hipMemcpy(d_B,B,k*n*sizeof(double), hipMemcpyHostToDevice));

	hipblasHandle_t handle;
	hipblasCreate(&handle);
//	cublasDGEMM(hipblasHandle_t handle, HIPBLAS_OP_N, HIPBLAS_OP_N,                         m, n, k,                           alpha,                           A, int lda,                           B, int ldb,                           beta,                           C, int ldc)

hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alphap, d_B, n, d_A, k, betap, d_C, n);

hipblasDestroy(handle);
hipMemcpy(C,d_C,m*n*sizeof(double),hipMemcpyDeviceToHost);
hipFree(d_A);
hipFree(d_B);
hipFree(d_C);
}



__global__
void cudaSeq(int m, int n, int k, double * A, double * B, double * C){

	for(int i = 0; i < n; i++){
		for(int j = 0;j < m;j++){
			C[i*n + j] = 0;
		}
	}

	for(int i = 0; i < m;i++){
		for(int l = 0; l < k;l++){
			for(int j = 0; j < n;j++){
				C[i*n + j] += A[i*k + l]*B[l*n + j];
			}
		}
	}

}

__global__
void cudaPar(int m, int n, int k, double * A, double * B, double * C){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;

	if(i < n && j < m){
		C[i + j*n] = 0;

		for(int l = 0; l < k;l++){
			C[j*n + i] += A[k*j + l]*B[l*n + i];
		}
	}
}

__global__
void cudaPar2(int m, int n, int k, int p, double * A, double * B, double * C){
//	printf("first in cudaPar2\n");
	const int P = 2;
	double C_r[P]={0.0,0.0};

//	int q = m%p;
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
//	printf("before if\n");

	j = j*p; // allows C[i + (0/1) j*n] indexing
	if(i < n && j < m-p + 1){ // only works if n%p = 0

		for(int pp = 0; pp < p; pp++){
			for(int l = 0; l < k; l++){
				C_r[pp] += A[(j+ pp)*k + l]*B[n*l + i];
			}
			C[(j + pp)*n + i] =  C_r[pp];
		}

	}

	if(i < n && j > m-p && j < m){
		for(int pp = 0; pp < p; pp++){
			for(int l = 0; l < k; l++){
				C_r[pp] += A[(j+ pp)*k + l]*B[n*l + i];
			}
			C[(j + pp)*n + i] =  C_r[pp];
		}


	}

}


__global__
void cudaPar4(int m, int n, int k, int p, double * A, double * B, double * C){
	const int P = 4;
	double C_r[P]={0.0,0.0,0.0,0.0};

//	int q = m%p;
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;

	j = j*p; // allows C[i + (0/1) j*n] indexing
	if(i < n && j < m-p + 1){
		for(int pp = 0; pp < p; pp++){
			for(int l = 0; l < k; l++){
				C_r[pp] += A[(j+ pp)*k + l]*B[n*l + i];
			}
			C[(j + pp)*n + i] =  C_r[pp];
		}
	}

	if(i < n && j > m-p && j < m){
		for(int pp = 0; pp < p; pp++){
			for(int l = 0; l < k; l++){
				C_r[pp] += A[(j+ pp)*k + l]*B[n*l + i];
			}
			C[(j + pp)*n + i] =  C_r[pp];
		}
	}

}

__global__
void cudaSMEM(int m, int n, int k, double * A, double * B, double * C){
	
	const int K = 16;	
	int kk = k/K;

	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	int iBlock = threadIdx.x;
	int jBlock = threadIdx.y;


	__shared__ double smemA[K][K];
	__shared__ double smemB[K][K];
	__shared__ double smemC[K][K];

	smemC[jBlock][iBlock] = 0;

	for(int q = 0; q < kk; q++){
		smemB[jBlock][iBlock] = B[i + q*K*n + jBlock*n];
		smemA[jBlock][iBlock] = A[iBlock + j*k + q*K];
		__syncthreads();
		for(int z = 0; z < K; z++){
			smemC[jBlock][iBlock] += smemA[jBlock][z]*smemB[z][iBlock];
		}
		__syncthreads();
	}
	C[i + j*n] = smemC[jBlock][iBlock];
}















// blocking
void matmult_blk(int m, int n, int k, double ** A, double ** B, double ** C, int bs) { 	
 	for(int i2 = 0; i2 < m;i2++){
		for(int j2 = 0; j2 < n;j2++){
			C[i2][j2] = 0;		
		}
 	
	} 
int bsi=bs;
int bsj=bs;
int bsl=bs;




for(int i1 = 0; i1 < m;i1+=bsi){
	if(m-i1 < bs) {bsi=m-i1;
	}
	for(int l1 = 0; l1 < k;l1+=bsl){
		if(k-l1 < bs) {bsl=k-l1;
		}
		for(int j1 = 0; j1 < n;j1+=bsj){
			if(n-j1 < bs) {bsj=n-j1;
			}
			for(int i2 = 0; i2 < bsi; i2++){	
				for(int l2 = 0; l2 < bsl;l2++){	
					for(int j2 = 0; j2 < bsj; j2++){	
							C[i1+i2][j1+j2] += A[i1+i2][l1+l2]*B[l1+l2][j1+j2];
					}
				}
			}
		}
	}
}
/*
for(int i2 = m1; i2 < m; i2++){	
	for(int l2 = n1; l2 < n;l2++){	
		for(int j2 = k1; j2 < k; j2++){	
			C[i2][j2] += A[i2][l2]*B[l2][j2];
			}
		}
	}
*/
} 


// Permutations of kmn
void matmult_kmn(int m, int n, int k, double ** A, double ** B, double ** C){
	for(int i = 0; i < m;i++){
		for(int j = 0; j < n;j++){
			C[i][j] = 0;		
		}
	}

for(int l = 0; l < k;l++){
	for(int i = 0; i < m;i++){
		for(int j = 0; j < n;j++){

				C[i][j] += A[i][l]*B[l][j];
			}
		}
	}
}

void matmult_knm(int m, int n, int k, double ** A, double ** B, double ** C){
	for(int i = 0; i < m;i++){
		for(int j = 0; j < n;j++){
			C[i][j] = 0;		
		}
	}

for(int l = 0; l < k;l++){
	for(int j = 0; j < n;j++){	
		for(int i = 0; i < m;i++){
				C[i][j] += A[i][l]*B[l][j];
			}
		}
	}
}


void matmult_mnk(int m, int n, int k, double ** A, double ** B, double ** C){
	for(int i = 0; i < m;i++){
		for(int j = 0; j < n;j++){
			C[i][j] = 0;		
		}
	}


	for(int i = 0; i < m;i++){
		for(int j = 0; j < n;j++){
			for(int l = 0; l < k;l++){
				C[i][j] += A[i][l]*B[l][j];
			}
		}
	}
}

void matmult_mkn(int m, int n, int k, double ** A, double ** B, double ** C){
	for(int i = 0; i < m;i++){
		for(int j = 0; j < n;j++){
			C[i][j] = 0;		
		}
	}


	for(int i = 0; i < m;i++){
		for(int l = 0; l < k;l++){
			for(int j = 0; j < n;j++){
				C[i][j] += A[i][l]*B[l][j];
			}
		}
	}
}

void matmult_nkm(int m, int n, int k, double ** A, double ** B, double ** C){
	for(int i = 0; i < m;i++){
		for(int j = 0; j < n;j++){
			C[i][j] = 0;		
		}
	}
for(int j = 0; j < n;j++){
	for(int l = 0; l < k;l++){
		for(int i = 0; i < m;i++){
				C[i][j] += A[i][l]*B[l][j];
			}
		}
	}
}

void matmult_nmk(int m, int n, int k, double ** A, double ** B, double ** C){
	for(int i = 0; i < m;i++){
		for(int j = 0; j < n;j++){
			C[i][j] = 0;		
		}
	}

	for(int j = 0; j < n;j++){
		for(int i = 0; i < m;i++){
			for(int l = 0; l < k;l++){
				C[i][j] += A[i][l]*B[l][j];
			}
		}
	}
}

__host__ __device__
void printMat(double *A, int m, int n){
	for(int i = 0; i < m;i++){
		for(int j = 0; j < n;j++){
			printf("%7.0f ",A[i*n + j]);
		}
		printf("\n");
	}	

}

// extern C
}
