#include "hip/hip_runtime.h"
#include "ass1_lib.h"
#include <math.h>
#include <algorithm>
extern "C" {
#include "cblas.h"
//}

// Nat
void matmult_nat(int m, int n, int k, double * A, double * B, double * C){
	for(int i = 0; i < m;i++){
		for(int j = 0; j < n;j++){
			C[i*n + j] = 0;
			for(int l = 0; l < k;l++){
				C[i*n + j] += A[i*k + l]*B[l*n + j];
			}
		}
	}
}

// library implementation through cblas
void matmult_lib(int m, int n, int k, double * A, double * B, double * C){
  cblas_dgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans, m, n, k, 1.0 , A, k, B, n, 0.0, C, n);
}

__host__
void matmult_gpu1(int m, int n, int k, double * A, double * B, double * C){
	
	double *d_A, *d_B, *d_C;
	hipMalloc(&d_A,n*m*sizeof(double));
	hipMalloc(&d_B,k*m*sizeof(double));
	hipMalloc(&d_C,n*k*sizeof(double));

 	hipMemcpy(d_A,A,n*m*sizeof(double), hipMemcpyHostToDevice);
	
//__global__	kernel()
	for(int i = 0; i < n; i++){
		for(int j = 0;j < m;j++){
			C[i*n + j] = 0;
		}
	}


	for(int i = 0; i < m;i++){
		for(int l = 0; l < k;l++){
			for(int j = 0; j < n;j++){
				C[i*n + j] += A[i*k + l]*B[l*n + j];
			}
		}
	}


}





















// blocking
void matmult_blk(int m, int n, int k, double ** A, double ** B, double ** C, int bs) { 	
 	for(int i2 = 0; i2 < m;i2++){
		for(int j2 = 0; j2 < n;j2++){
			C[i2][j2] = 0;		
		}
 	
	} 
int bsi=bs;
int bsj=bs;
int bsl=bs;




for(int i1 = 0; i1 < m;i1+=bsi){
	if(m-i1 < bs) {bsi=m-i1;
	}
	for(int l1 = 0; l1 < k;l1+=bsl){
		if(k-l1 < bs) {bsl=k-l1;
		}
		for(int j1 = 0; j1 < n;j1+=bsj){
			if(n-j1 < bs) {bsj=n-j1;
			}
			for(int i2 = 0; i2 < bsi; i2++){	
				for(int l2 = 0; l2 < bsl;l2++){	
					for(int j2 = 0; j2 < bsj; j2++){	
							C[i1+i2][j1+j2] += A[i1+i2][l1+l2]*B[l1+l2][j1+j2];
					}
				}
			}
		}
	}
}
/*
for(int i2 = m1; i2 < m; i2++){	
	for(int l2 = n1; l2 < n;l2++){	
		for(int j2 = k1; j2 < k; j2++){	
			C[i2][j2] += A[i2][l2]*B[l2][j2];
			}
		}
	}
*/
} 


// Permutations of kmn
void matmult_kmn(int m, int n, int k, double ** A, double ** B, double ** C){
	for(int i = 0; i < m;i++){
		for(int j = 0; j < n;j++){
			C[i][j] = 0;		
		}
	}

for(int l = 0; l < k;l++){
	for(int i = 0; i < m;i++){
		for(int j = 0; j < n;j++){

				C[i][j] += A[i][l]*B[l][j];
			}
		}
	}
}

void matmult_knm(int m, int n, int k, double ** A, double ** B, double ** C){
	for(int i = 0; i < m;i++){
		for(int j = 0; j < n;j++){
			C[i][j] = 0;		
		}
	}

for(int l = 0; l < k;l++){
	for(int j = 0; j < n;j++){	
		for(int i = 0; i < m;i++){
				C[i][j] += A[i][l]*B[l][j];
			}
		}
	}
}


void matmult_mnk(int m, int n, int k, double ** A, double ** B, double ** C){
	for(int i = 0; i < m;i++){
		for(int j = 0; j < n;j++){
			C[i][j] = 0;		
		}
	}


	for(int i = 0; i < m;i++){
		for(int j = 0; j < n;j++){
			for(int l = 0; l < k;l++){
				C[i][j] += A[i][l]*B[l][j];
			}
		}
	}
}

void matmult_mkn(int m, int n, int k, double ** A, double ** B, double ** C){
	for(int i = 0; i < m;i++){
		for(int j = 0; j < n;j++){
			C[i][j] = 0;		
		}
	}


	for(int i = 0; i < m;i++){
		for(int l = 0; l < k;l++){
			for(int j = 0; j < n;j++){
				C[i][j] += A[i][l]*B[l][j];
			}
		}
	}
}

void matmult_nkm(int m, int n, int k, double ** A, double ** B, double ** C){
	for(int i = 0; i < m;i++){
		for(int j = 0; j < n;j++){
			C[i][j] = 0;		
		}
	}
for(int j = 0; j < n;j++){
	for(int l = 0; l < k;l++){
		for(int i = 0; i < m;i++){
				C[i][j] += A[i][l]*B[l][j];
			}
		}
	}
}

void matmult_nmk(int m, int n, int k, double ** A, double ** B, double ** C){
	for(int i = 0; i < m;i++){
		for(int j = 0; j < n;j++){
			C[i][j] = 0;		
		}
	}

	for(int j = 0; j < n;j++){
		for(int i = 0; i < m;i++){
			for(int l = 0; l < k;l++){
				C[i][j] += A[i][l]*B[l][j];
			}
		}
	}
}
}
