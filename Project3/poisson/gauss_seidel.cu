
#include <hip/hip_runtime.h>
void gauss_seidel(double ** u, double ** f, int N, double delta2){

	int i,j;
	for(i = 1; i < N-1; i++){
		for(j = 1; j < N-1; j++){
			u[i][j] = 0.25*(u[i-1][j] + u[i+1][j] + u[i][j-1] + u[i][j+1] + delta2*f[i][j]);
		}
	}
}
