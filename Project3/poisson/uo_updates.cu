
#include <hip/hip_runtime.h>
__global__ void update_uo_multi_kernel0(double * d0_u, double * d0_uo, int N){
	int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
	int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
	
	d0_uo[i*N + j] = d0_u[i*N + j];
}

__global__ void update_uo_multi_kernel1(double * d1_u, double * d1_uo, int N){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
	d1_uo[i*N + j] = d1_u[i*N + j];
}
