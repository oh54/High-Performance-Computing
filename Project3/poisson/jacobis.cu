#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
__global__ 
void jacobi_seq_kernel(double * d_u, double * d_uo, double * d_f, int N, double delta2){
	int i,j;
	for(i = 1; i < N-1; i++){
		for(j = 1; j < N-1; j++){
			d_u[i*N + j] = 0.25*(d_uo[(i-1)*N + j] + d_uo[(i+1)*N + j] + d_uo[i*N + j+1] + d_uo[i*N + j-1] + delta2*d_f[i*N + j]);
		}
	}
}

__global__ 
void jacobi_single_kernel(double * d_u, double * d_uo, double * d_f, int N, double delta2){
	int j = blockIdx.x * blockDim.x + threadIdx.x + 1;
	int i = blockIdx.y * blockDim.y + threadIdx.y + 1;
	d_u[i*N + j] = 0.25*(d_uo[(i-1)*N + j] + d_uo[(i+1)*N + j] + d_uo[i*N + j+1] + d_uo[i*N + j-1] + delta2*d_f[i*N + j]);
}

__global__ 
void jacobi_multi_kernel(double * d0_u, double * d0_uo, double * d0_f, int N, double delta2){
	int j = blockIdx.x * blockDim.x + threadIdx.x + 1;
	int i = blockIdx.y * blockDim.y + threadIdx.y + 1;
	
	d0_u[i*N + j] = 0.25*(d0_uo[(i-1)*N + j] + d0_uo[(i+1)*N + j] + d0_uo[i*N + j+1] + d0_uo[i*N + j-1] + delta2*d0_f[i*N + j]);
}




