#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
__global__ 
void jacobi_seq_kernel(double * d_u, double * d_uo, double * d_f, int N, double delta2){

	int i,j;
	for(i = 1; i < N-1; i++){
		for(j = 1; j < N-1; j++){
			d_u[i*N + j] = 0.25*(d_uo[(i-1)*N + j] + d_uo[(i+1)*N + j] + d_uo[i*N + j+1] + d_uo[i*N + j-1] + delta2*d_f[i*N + j]);
		}
	}
}

__global__ 
void jacobi_single_kernel(double * d_u, double * d_uo, double * d_f, int N, double delta2){
	int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
	int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
	d_u[i*N + j] = 0.25*(d_uo[(i-1)*N + j] + d_uo[(i+1)*N + j] + d_uo[i*N + j+1] + d_uo[i*N + j-1] + delta2*d_f[i*N + j]);
}

__global__ 
void jacobi_multi_kernel0(double * d_u, double * d_uo, double * d_f, int N, double delta2){
	int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
	int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
	//printf("I AND J 0 (%i, %i)\n", i, j);	
	d_u[i*N + j] = 0.25*(d_uo[(i-1)*N + j] + d_uo[(i+1)*N + j] + d_uo[i*N + j+1] + d_uo[i*N + j-1] + delta2*d_f[i*N + j]);
}

__global__ 
void jacobi_multi_kernel1(double * d_u, double * d_uo, double * d_f, int N, double delta2){
	int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	//printf("I AND J 1 (%i, %i)\n", i, j);	
	d_u[i*N + j] = 0.25*(d_uo[(i-1)*N + j] + d_uo[(i+1)*N + j] + d_uo[i*N + j+1] + d_uo[i*N + j-1] + delta2*d_f[i*N + j]);
}


