#include "hip/hip_runtime.h"
#include "stdio.h"
#include "stdlib.h"
#include <math.h>
#include <time.h>
#include <sys/time.h>
#include <string.h>
#include "ass2_lib.h"
#include <omp.h>
#include "hip/hip_runtime_api.h"


void printMat(double * A, int N){
	int i,j;
	for(i = 0; i < N; i++){
		for(j = 0; j < N; j++){
			printf("%7.0f ",A[i*N + j]);
		}	
		printf("\n");
	}
}

double getMatSum(double * A, int N){
	int i,j;
	double sum = 0.0;
	for(i = 0; i < N; i++){
		for(j = 0; j < N; j++){
			sum += A[i*N + j];
		}	
	}
	return sum;
}



double fnorm_squared(double * u, double * uo, int N){
	int i,j;
	double sum = 0;
	for(i = 1; i <N-1; i++){
		for(j = 1; j<N-1; j++){
			sum += (u[i*N + j]-uo[i*N + j])*(u[i*N + j]-uo[i*N + j]);
		}
	}
	return sum / (N*N);
}

void update_uo(double * u, double * uo, int N){
	int i,j;
	for(i = 0; i<N; i++){
		for(j = 0; j<N; j++){
			uo[i*N + j] = u[i*N + j];
		}
	}
}


void initialize_matrices(double * u, double * uo, double * f, int N, double Nt){

	
	// init loop variables
	int i, j;

	// define uo as zeros\
	uo[x][0] = 0 i.e. outer wall
	for(i = 1; i < N; i++){
		for(j = 1; j < N-1; j++){
			uo[i*N + j] = 0;
		}
	}
	
	// Defining the boundaries to 20
	{
		for(j = 0; j < N; j++) uo[j] = 20;
		for(i = 0; i < N; i++) uo[i*N] = 20;
		for(i = 0; i < N; i++) uo[i*N + N-1] = 20;
	}
		
	//setting u = uo;
	for(i = 0; i<N; i++){
		for(j = 0; j<N; j++){
			u[i*N + j] = uo[i*N + j];
		}
	}
	//printMat(uo,N);

	// defining the f matrix
	for(i = 0; i < N; i++){
		for(j = 0; j < N; j++){
			f[i*N + j] = 0;
		}
	}

	for(i = 4*Nt; i < 5*Nt; i++){
		for(j = 3*Nt; j < 4*Nt; j++){
			f[i*N + j] = 200;
		}
	}
}
__global__
void ChangePointers(double ** p1, double ** p2) 
{
    double * temp = *p1;
    *p1 = *p2;
    *p2 = temp;
}

__global__
void PrintPointers(double * d_u, double * d_uo){
	printf("&D_U: %i\n", d_u);
	printf("&D_UO: %i\n", d_uo);
}



__host__
void doSeq(double * u, double * uo, double * f, int N, double d, int kmax, double delta2, double dd){
	double start = omp_get_wtime(); 
	double *d_u, *d_uo, *d_f;
	int memsize = N*N*sizeof(double);
	hipMalloc(&d_u, memsize);
	hipMalloc(&d_uo, memsize);
	hipMalloc(&d_f, memsize);	
	hipMemcpy(d_u, u, memsize, hipMemcpyHostToDevice);
	hipMemcpy(d_uo, uo, memsize, hipMemcpyHostToDevice);
	hipMemcpy(d_f, f, memsize, hipMemcpyHostToDevice);
	int k = 0;
	double checksum = 1000.0;
	while(k < kmax){
		jacobi_seq_kernel<<<1, 1>>>(d_u, d_uo, d_f, N, delta2);
		double * temp = d_uo;
    		d_uo = d_u;
    		d_u = temp;
		k++;
	}

	hipMemcpy(uo, d_uo, memsize, hipMemcpyDeviceToHost);
	printf("%s, ", "CU-SEQ");
	printf("%f, ", omp_get_wtime()-start);
	printf("%i, %.20f, %.0f, %i\n", N, dd, getMatSum(uo, N), k);
	hipFree(d_u);
	hipFree(d_uo);
	hipFree(d_f);
}


__host__
void doSingle(double * u, double * uo, double * f, int N, double d, int kmax, double delta2, double dd){
	double *d_u, *d_uo, *d_f;
	int memsize = N*N*sizeof(double);
	hipMalloc(&d_u, memsize);
	hipMalloc(&d_uo, memsize);
	hipMalloc(&d_f, memsize);	
	hipMemcpy(d_u, u, memsize, hipMemcpyHostToDevice);
	hipMemcpy(d_uo, uo, memsize, hipMemcpyHostToDevice);
	hipMemcpy(d_f, f, memsize, hipMemcpyHostToDevice);
	int k = 0;
	double checksum = 1000.0;	
	hipSetDevice(6);
	int K = 16;
	int gridx = ceil((N-2)*1.0/(K));
	int gridy = ceil((N-2)*1.0/(K));
	double start = omp_get_wtime(); 
	while(k < kmax){
		jacobi_single_kernel<<<dim3(gridx,gridy),dim3(K,K)>>>(d_u, d_uo, d_f, N, delta2);
		double * temp = d_uo;
    		d_uo = d_u;
    		d_u = temp;
		k++;
	}
	double end = omp_get_wtime(); 
	hipMemcpy(uo, d_uo, memsize, hipMemcpyDeviceToHost);

	//printf("MATRIX UO:\n");
	//printMat(uo,N);
	//printf("\n");

	printf("%s, ", "CU-SIN");
	printf("%f, ", end-start);
	printf("%i, %.20f, %i, %.0f\n", N, dd, k, getMatSum(uo, N));
	hipFree(d_u);
	hipFree(d_uo);
	hipFree(d_f);


}


__host__
void doMulti(double * u, double * uo, double * f, int N, double d, int kmax, double delta2, double dd){
	
	double *d0_u, *d0_uo, *d0_f, *d1_u, *d1_uo, *d1_f;
	int memsize = N*N*sizeof(double);
	int Nsize = N*sizeof(double);

	hipSetDevice(6);
	hipDeviceEnablePeerAccess(7,0);
	hipMalloc((void**)&d0_u, memsize/2 + Nsize);
	hipMalloc((void**)&d0_uo, memsize/2 + Nsize);
	hipMalloc((void**)&d0_f, memsize/2 + Nsize);
	hipMemcpy(d0_u, u, memsize/2 + Nsize, hipMemcpyHostToDevice);
	hipMemcpy(d0_uo, uo, memsize/2 + Nsize, hipMemcpyHostToDevice);
	hipMemcpy(d0_f, f, memsize/2 + Nsize, hipMemcpyHostToDevice);

	hipSetDevice(7); 
	hipDeviceEnablePeerAccess(6,0);
	hipMalloc((void**)&d1_u, memsize/2 + Nsize);
	hipMalloc((void**)&d1_uo, memsize/2 + Nsize);
	hipMalloc((void**)&d1_f, memsize/2 + Nsize);
	hipMemcpy(d1_u, &u[memsize/2/sizeof(double) -N], memsize/2 + Nsize, hipMemcpyHostToDevice);
	hipMemcpy(d1_uo, &uo[memsize/2/sizeof(double) -N], memsize/2 + Nsize, hipMemcpyHostToDevice);
	hipMemcpy(d1_f, &f[memsize/2/sizeof(double) -N], memsize/2 + Nsize, hipMemcpyHostToDevice);
	int k = 0;
	double checksum = 1000.0;	
	int K = 16;
	int gridx = ceil((N-2)*1.0/(K));
 	int gridy = ceil((N-2)*1.0/(K));
	gridy = ceil(gridy*1.0 / 2);
	double start = omp_get_wtime(); 
	while(k < kmax){
		hipSetDevice(6);
		jacobi_multi_kernel<<<dim3(gridx,gridy),dim3(K,K)>>>(d0_u, d0_uo, d0_f, N, delta2);
		hipSetDevice(7);
		jacobi_multi_kernel<<<dim3(gridx,gridy),dim3(K,K)>>>(d1_u, d1_uo, d1_f, N, delta2);
		hipDeviceSynchronize();
		double * temp = d0_uo;
    		d0_uo = d0_u;
    		d0_u = temp;
		double * temp2 = d1_uo;
		d1_uo = d1_u;
    		d1_u = temp2;
		hipMemcpy(d1_uo, d0_uo+(N-2)/2*N, Nsize, hipMemcpyDeviceToDevice);
		hipMemcpy(d0_uo+(N-2)/2*N+N , d1_uo+N, Nsize, hipMemcpyDeviceToDevice);
		k++;
	}
	double end = omp_get_wtime();
	hipSetDevice(6);
	hipMemcpy(uo, d0_uo, memsize/2, hipMemcpyDeviceToHost);
	hipSetDevice(7);
	hipMemcpy(&uo[memsize/2/sizeof(double)], &d1_uo[N], memsize/2, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	//printf("MATRIX UO:\n");
	//printMat(uo,N);
	//printf("\n");

	printf("%s, ", "CU-MUL");
	printf("%f, ", end-start);
	printf("%i, %.20f, %i, %.0f\n", N, dd, k, getMatSum(uo, N));
	hipFree(d1_u);
	hipFree(d1_uo);
	hipFree(d1_f);
	hipFree(d0_u);
	hipFree(d0_uo);
	hipFree(d0_f);

	
}





int main(int argc, char **argv){
	// ./poisson <method type> <NN> <d> <kmax>
	
	int NN;
	double dd;
	int kmax;
	
	sscanf(argv[2] , "%d", &NN);
	sscanf(argv[3] , "%lf", &dd);
	sscanf(argv[4] , "%d", &kmax);

	double d = dd*dd;
	int N = NN + 2;
	double delta = 2.0/N;
	double delta2 = delta*delta; 
	double Nt = N/6.0; 	

	double * u, * uo, * f;

	u = (double*)malloc(N*N*sizeof(double));
	uo = (double*)malloc(N*N*sizeof(double));
	f = (double*)malloc(N*N*sizeof(double));

	initialize_matrices(u,uo,f, N,Nt);
	//printf("MATRIX U:\n");
	//printMat(u,N);
	//printf("\n");
	//printf("MATRIX UO:\n");
	//printMat(uo,N);
	//printf("\n");

	if(strcmp(argv[1], "seq") == 0){
		doSeq(u, uo, f, N, d, kmax, delta2, dd); 
	}

	
	// naive single GPU
	// NN must be multiple of K
	if(strcmp(argv[1], "sin") == 0){
		doSingle(u, uo, f, N, d, kmax, delta2, dd);

	}
	
	
	// naive multi GPU
	// NN must be even multiple of K
	if(strcmp(argv[1], "mul") == 0){
		doMulti(u, uo, f, N, d, kmax, delta2, dd);



	}
	
}















// WEEK2 OPENMP STUFF
/*
	//initialize_matrices(u, uo, f, N, Nt);
	int i, j;
	struct timeval  tv1, tv2;
	double runtime;
	int nruns;

	if(strcmp(argv[1], "jacobi") == 0){

		runtime = 0.0;
		nruns = 0;
		
		while(runtime <= 3.0){
			k = 0;
			checksum = 1000;
			initialize_matrices(u, uo, f, N, Nt);			

			gettimeofday(&tv1, NULL);
			while(checksum > d && k < kmax){
				jacobi_seq(u,uo,f,N,delta2);
				checksum = fnorm_squared(u,uo,N);
				for(i = 0; i<N; i++){
					for(j = 0; j<N; j++){
						uo[i][j] = u[i][j];
					}
				}
				k++;
//				printf("%f \n", checksum);
			}
			gettimeofday(&tv2, NULL);
			runtime += (double) (tv2.tv_usec - tv1.tv_usec) / 1000000 + (double) (tv2.tv_sec - tv1.tv_sec);
			nruns++;
		}
		printf("%s, ", "JAC");
		printf("%f, ", (double) runtime / nruns);
		printf("%i, %.20f, %i, %i\n", N, dd, k, k*nruns);
		
	}

	if(strcmp(argv[1], "gauss") == 0){

		runtime = 0.0;
		nruns = 0;

		while(runtime <= 3.0){
			k = 0;
			checksum = 1000;
			initialize_matrices(u, uo, f, N, Nt);			
			
			gettimeofday(&tv1, NULL);
			while(checksum > d && k < kmax){
				gauss_seidel(u,f,N,delta2);
				checksum = fnorm_squared(u,uo,N);
				for(i = 0; i<N; i++){
					for(j = 0; j<N; j++){
						uo[i][j] = u[i][j];
					}
				}
				k++;
			}
			gettimeofday(&tv2, NULL);
			runtime += (double) (tv2.tv_usec - tv1.tv_usec) / 1000000 + (double) (tv2.tv_sec - tv1.tv_sec);
			nruns++;
		}
		
		printf("%s, ", "G-S");
		printf("%f, ", runtime);
		printf("%i, %.20f, %i, %i\n", N, dd, k, k*nruns);
		
	}
	
	if(strcmp(argv[1], "omp") == 0){
		runtime = 0.0;
	
		k = 0;
		checksum = 1000;
		initialize_matrices(u, uo, f, N, Nt);			
		
		double omp_s = omp_get_wtime();
		while(checksum > d && k < kmax){
			#pragma omp parallel default(none) shared(u,uo,f,N,delta2) private(i,j)
			{
				jacobi_seq(u,uo,f,N,delta2);
			} // end parallel
			checksum = fnorm_squared(u,uo,N);
			for(i = 0; i<N; i++){
				for(j = 0; j<N; j++){
					uo[i][j] = u[i][j];
				}
			} 
			
			k++;
		}
		double omp_time = omp_get_wtime() - omp_s;
		int thread = omp_get_max_threads();
		printf("%s, ", "OMP");
		printf("%f, ", omp_time);
		printf("%i, %.20f, %i, %i\n", N, dd, k, thread);

	}


	if(strcmp(argv[1], "omp2") == 0){
			runtime = 0.0;
	
		k = 0;
		checksum = 1000;
		initialize_matrices(u, uo, f, N, Nt);			
		
		double omp_s = omp_get_wtime();
			#pragma omp parallel default(none) shared	(u,uo,f,N,delta2, checksum, k, d, kmax) private(i,j)
				{
			while(checksum > d && k < kmax){
					
					jacobi_seq(u,uo,f,N,delta2);
					
					// checksum = fnorm_squared(u,uo,N);
					#pragma omp for	private(i,j)  reduction(+:checksum)
					for(i = 1; i <N-1; i++){
						for(j = 1; j<N-1; j++){
							checksum += (u[i][j]-uo[i][j])*(u[i][j]-uo[i][j]);
						}
					}
					
					
					#pragma omp for	private(i,j) 
					for(i = 0; i<N; i++){
						for(j = 0; j<N; j++){
							uo[i][j] = u[i][j];
						}
					} 
					#pragma omp master
					{
					k++;
					checksum=checksum/(N*N);		
					}
					#pragma omp barrier
				} // end while 
				
			} // end parallel
		double omp_time = omp_get_wtime() - omp_s;
		int thread = omp_get_max_threads();
		printf("%s, ", "OMP2");
		printf("%f, ", omp_time);
		printf("%i, %.20f, %i, %i\n", N, dd, k, thread);

	}

	if(strcmp(argv[1], "omp3") == 0){
		runtime = 0.0;
		k = 0;
		checksum = 1000;
		#pragma omp parallel default(none) shared(u, uo, f, N, Nt)
		{
			initialize_matrices(u, uo, f, N, Nt);			
		}
		double omp_s = omp_get_wtime();
			#pragma omp parallel default(none) shared	(u,uo,f,N,delta2, checksum, k, d, kmax) private(i,j)
			{
				while(checksum > d && k < kmax){
					jacobi_seq(u,uo,f,N,delta2);
					// checksum = fnorm_squared(u,uo,N);
					#pragma omp for	private(i,j)  reduction(+:checksum)
					for(i = 1; i <N-1; i++){
						for(j = 1; j<N-1; j++){
							checksum += (u[i][j]-uo[i][j])*(u[i][j]-uo[i][j]);
						}
					}
					#pragma omp for	private(i,j) 
					for(i = 0; i<N; i++){
						for(j = 0; j<N; j++){
							uo[i][j] = u[i][j];
						}
					} 
					#pragma omp master
					{
					k++;
					checksum=checksum/(N*N);
					}
					#pragma omp barrier
				} // end while 
			} // end parallel
	double omp_time = omp_get_wtime() - omp_s;
	int thread = omp_get_max_threads();
	printf("%s, ", "OMP3");
	printf("%f, ", omp_time);
	printf("%i, %.20f, %i, %i\n", N, dd, k, thread);

	}

	

//	printMat(u,N);
	// Save the data

	/*
	The real code should be here. While loop that checks if change from uo to u is small enough to be accepted (solution has converged). Jacobi should be implemented as a sub-routine in a separate function
	*/

	//printf("k is: %i \n",k);

//	dfree_2d(u);
//	dfree_2d(uo);
//	dfree_2d(f);
//}

