#include "hip/hip_runtime.h"
#include "stdio.h"
#include "stdlib.h"
#include "hip/hip_runtime_api.h"
__global__ void helloWorldKernelFunc(void){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int max_i = gridDim.x * blockDim.x;

  if (i == 100){
    int *a = (int*) 0x10000; *a = 0;
  }

  printf("Hello World! I'm thread %i out of %i in block %i. My global thread id is %i out of %i\n", threadIdx.x, blockDim.x, blockIdx.x, i, max_i);
}

int main(int argc, char **argv){
  //Allocate memory space on host and device
  //h_data = malloc(...);
  //hipMalloc(...);
  //Transfer data from host to device
  //hipMemcpy(...);
  // Kernel lauch
  helloWorldKernelFunc<<<4, 64>>>();
  checkCudaErrors(hipDeviceSynchronize());
  // Transfer results from device to host
  //hipMemcpy(...);
  // Free memory
  //free(h_data);
  //hipFree(...);
} 
